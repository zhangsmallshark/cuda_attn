#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Standard Library includes
#include <iostream>
#include <vector>

#include "cutlass/gemm/device/gemm.h"

#include "cutlass/cutlass.h"
#include "cutlass/util/command_line.h"
#include "cutlass/util/host_tensor.h"

/// Define a CUTLASS GEMM template and launch a GEMM kernel.
template <typename LayoutInputA, typename LayoutInputB>
hipError_t CutlassSgemm(int M, int N, int K, float alpha, float const *A,
                         int lda, float const *B, int ldb, float beta, float *C,
                         int ldc, float *bias, hipStream_t stream) {

  // Define type definition for single-precision CUTLASS GEMM with column-major
  // input matrices and 128x128x8 threadblock tile size (chosen by default).
  //
  // To keep the interface manageable, several helpers are defined for plausible
  // compositions including the following example for single-precision GEMM.
  // Typical values are used as default template arguments. See
  // `cutlass/gemm/device/default_gemm_configuration.h` for more details.
  //
  // To view the full gemm device API interface, see
  // `cutlass/gemm/device/gemm.h`

  //   using ColumnMajor = cutlass::layout::ColumnMajor;

  //   using CutlassGemm = cutlass::gemm::device::Gemm<float,        //
  //   Data-type of A matrix
  //                                                   ColumnMajor,  // Layout
  //                                                   of A matrix float, //
  //                                                   Data-type of B matrix
  //                                                   ColumnMajor,  // Layout
  //                                                   of B matrix float, //
  //                                                   Data-type of C matrix
  //                                                   ColumnMajor>; // Layout
  //                                                   of C matrix

  ///////////////////////////////////////////////////////////////////////////////////////////////////

  // The code section below describes datatype for input, output matrices and
  // computation between elements in input matrices.
  using ElementAccumulator = float; // <- data type of accumulator
  using ElementComputeEpilogue =
      ElementAccumulator;      // <- data type of epilogue operations
  using ElementInputA = float; // <- data type of elements in input matrix A
  using ElementInputB = float; // <- data type of elements in input matrix B
  using ElementOutput = float; // <- data type of elements in output matrix D

  // The code section below describes matrix layout of input and output
  // matrices. Column Major for Matrix A, Row Major for Matrix B and Row Major
  // for Matrix C using LayoutInputA = cutlass::layout::RowMajor; using
  // LayoutInputB = cutlass::layout::ColumnMajor; using LayoutOutput =
  // cutlass::layout::RowMajor;

  // using LayoutInputA =
  //     cutlass::layout::RowMajor; // (trans_a)? cutlass::layout::ColumnMajor :
  //                                // cutlass::layout::RowMajor;
  // using LayoutInputB =
  //     cutlass::layout::ColumnMajor; // (trans_b)?
  //     cutlass::layout::ColumnMajor :
  //                                   // cutlass::layout::RowMajor;
  using LayoutOutput = cutlass::layout::RowMajor;

  // This code section describes whether you want to use tensor cores or regular
  // SIMT cores on GPU SM
  using MMAOp = cutlass::arch::OpClassSimt;

  // This code section describes CUDA SM architecture number
  using SmArch = cutlass::arch::Sm80;

  // This code section describes the tile size a thread block will compute
  using ShapeMMAThreadBlock =
      cutlass::gemm::GemmShape<256, 128, 16>; // <- threadblock tile M = 256, N
                                              // = 128, K = 16
  // This code section describes tile size a warp will compute
  using ShapeMMAWarp =
      cutlass::gemm::GemmShape<64, 64,
                               16>; // <- warp tile M = 64, N = 64, K = 16
  // This code section describes the size of MMA op
  using ShapeMMAOp =
      cutlass::gemm::GemmShape<1, 1, 1>; // <- MMA Op tile M = 16, N = 8, K = 8

  // This code section describes how threadblocks are scheduled on GPU
  using SwizzleThreadBlock =
      cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>; // <- ??

  // This code section describes the epilogue part of the kernel
  using EpilogueOp = cutlass::epilogue::thread::LinearCombination<
      ElementOutput, // <- data type of output matrix
      1, // 128 / cutlass::sizeof_bits<ElementOutput>::value,  // <- the number
         // of elements per vectorized
         //  memory access. For a byte, it's 16
         //  elements. This becomes the vector width of
         //  math instructions in the epilogue too
      ElementAccumulator,      // <- data type of accumulator
      ElementComputeEpilogue>; // <- data type for alpha/beta in linear
                               // combination function

  // Number of pipelines you want to use
  constexpr int NumStages = 4;

  using CutlassGemm = cutlass::gemm::device::Gemm<
      ElementInputA, LayoutInputA, ElementInputB, LayoutInputB, ElementOutput,
      LayoutOutput, ElementAccumulator, MMAOp, SmArch, ShapeMMAThreadBlock,
      ShapeMMAWarp, ShapeMMAOp, EpilogueOp, SwizzleThreadBlock, NumStages>;

  // Define a CUTLASS GEMM type
  CutlassGemm gemm_operator;

  // Construct the CUTLASS GEMM arguments object.
  //
  // One of CUTLASS's design patterns is to define gemm argument objects that
  // are constructible in host code and passed to kernels by value. These may
  // include pointers, strides, scalars, and other arguments needed by Gemm and
  // its components.
  //
  // The benefits of this pattern are (1.) a structured, composable strategy for
  // passing host-constructible arguments to kernels and (2.) minimized
  // initialization overhead on kernel entry.
  //

  float* c_ptr = bias? bias : C;
  int ldc_b = bias? 0 : ldc;

  typename CutlassGemm::Arguments args(
      {M, N, K}, // Gemm Problem dimensions
      {A, lda},  // Tensor-ref for source matrix A
      {B, ldb},  // Tensor-ref for source matrix B
      {c_ptr, ldc_b}, // Tensor-ref for source matrix C
      {C, ldc},  // Tensor-ref for destination matrix D (may be different memory
                 // than source C matrix)
      {alpha, beta}); // Scalars used in the Epilogue


  // Using the arguments, query for extra workspace required for matrix
  // multiplication computation
  size_t workspace_size = CutlassGemm::get_workspace_size(args);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  //
  // Launch the CUTLASS GEMM kernel.
  //

  cutlass::Status status = gemm_operator(args, workspace.get(), stream);

  //
  // Return a hipError_t if the CUTLASS GEMM operator returned an error code.
  //

  if (status != cutlass::Status::kSuccess) {
    return hipErrorUnknown;
  }

  // Return success, if no errors were encountered.
  return hipSuccess;
}


std::vector<torch::Tensor> linear_cutlass_forward(torch::Tensor input,
                                                  torch::Tensor weights,
                                                  torch::Tensor bias) {
  // input: (batch_size, in_features)
  // weights: (out_features, in_features)
  // bias : (out_features)

  auto in_sizes = input.sizes();
  int batch_size = in_sizes[0];
  int in_features = in_sizes[1];
  int out_features = weights.size(0);

  float alpha = 1.0;
  float beta = 0;

  // Compute leading dimensions for each matrix.
  int lda = in_features;
  int ldb = in_features;
  int ldc = out_features;

  auto one_vec = torch::ones(batch_size);

  hipStream_t stream0;
  hipStreamCreate(&stream0);

  // output = input * weights^T (without biases)
  // auto output = torch::zeros_like(input);
  auto output =
      torch::empty({batch_size, out_features},
                   torch::dtype(torch::kFloat32).device(torch::kCUDA, 0));
  CutlassSgemm<cutlass::layout::RowMajor, cutlass::layout::ColumnMajor>(
      batch_size, out_features, in_features, alpha, input.data_ptr<float>(),
      lda, weights.data_ptr<float>(), ldb, beta, output.data_ptr<float>(), ldc,
      bias.data_ptr<float>(), stream0);

  // output += biases * one_vec^T
  // float beta = 1.0;
  // result = CutlassSgemm(M, N, K, alpha, bias.data_ptr<float>(), lda,
  // one_vec.data_ptr<float>(), ldb, beta, output.data_ptr<float>(), ldc,
  // stream0);

  return {output};
}

std::vector<torch::Tensor> linear_cutlass_backward(torch::Tensor grad_output,
                                                   torch::Tensor input,
                                                   torch::Tensor weights) {
  auto in_sizes = input.sizes();
  int batch_size = in_sizes[0];
  int in_features = in_sizes[1];
  int out_features = weights.size(0);

  auto grad_weights = torch::zeros_like(weights); // (out_features, in_features)
  auto grad_input = torch::zeros_like(input);     // (batch_size, in_features)

  // int M = 1024;
  // int K = 1024;
  // int N = 1024;

  // int lda = M;
  // int ldb = K;
  // int ldc = M;

  float alpha = 1.0;
  float beta = 0;
  // int batch_size = 1024;
  auto grad_biases = torch::sum(grad_output, 0);

  hipStream_t stream0;
  hipStreamCreate(&stream0);

  // db = (dy) * one_vec
  // result = CutlassSgemm(M, N, K, alpha, grad_output.data_ptr<float>(), lda,
  // one_vec.data_ptr<float>(), ldb, beta, grad_biases, ldc, stream0);

  // dw = (dy)^T * x
  CutlassSgemm<cutlass::layout::ColumnMajor, cutlass::layout::RowMajor>(
      out_features, in_features, batch_size, alpha,
      grad_output.data_ptr<float>(), out_features, input.data_ptr<float>(),
      in_features, beta, grad_weights.data_ptr<float>(), in_features, nullptr, stream0);

  // dx = dy * w
  CutlassSgemm<cutlass::layout::RowMajor, cutlass::layout::RowMajor>(
      batch_size, in_features, out_features, alpha,
      grad_output.data_ptr<float>(), out_features, weights.data_ptr<float>(),
      in_features, beta, grad_input.data_ptr<float>(), in_features, nullptr, stream0);

  return {grad_input, grad_weights, grad_biases};
}
