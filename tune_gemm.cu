#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*
  This example demonstrates how to call a CUTLASS GEMM kernel and provides a naive reference
  matrix multiply kernel to verify its correctness.

  The CUTLASS Gemm template is instantiated in the function CutlassSgemmNN. This is kernel computes
  the general matrix product (GEMM) using single-precision floating-point arithmetic and assumes
  all matrices have column-major layout.

  The threadblock tile size is chosen as 128x128x8 which offers good performance for large matrices.
  See the CUTLASS Parallel for All blog post for more exposition on the tunable parameters available
  in CUTLASS.

  https://devblogs.nvidia.com/cutlass-linear-algebra-cuda/

  Aside from defining and launching the SGEMM kernel, this example does not use any other components
  or utilities within CUTLASS. Such utilities are demonstrated elsewhere in other examples and are
  prevalent in the CUTLASS unit tests.

  This example has delibrately been kept similar to the basic_gemm example from cutlass-1.3 to
  highlight the minimum amount of differences needed to transition to cutlass-2.0.

  Cutlass-1.3 sgemm: https://github.com/NVIDIA/cutlass/blob/master/examples/00_basic_gemm/basic_gemm.cu
*/

// Standard Library includes
#include <iostream>
#include <sstream>
#include <vector>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "hip/hip_runtime_api.h"

// Helper methods to check for errors
#include "helper.h"

//
// CUTLASS includes needed for single-precision GEMM kernel
//

// Defines cutlass::gemm::device::Gemm, the generic Gemm computation template class.
#include "cutlass/gemm/device/gemm.h"

#include "cutlass/cutlass.h"
#include "cutlass/util/command_line.h"
#include "cutlass/util/host_tensor.h"
///////////////////////////////////////////////////////////////////////////////////////////////////
//
// This function defines a CUTLASS GEMM kernel instantiation, constructs its parameters object,
// and launches it on the CUDA device.
//
///////////////////////////////////////////////////////////////////////////////////////////////////

/// Define a CUTLASS GEMM template and launch a GEMM kernel.
hipError_t CutlassSgemmNN(
  int M,
  int N,
  int K,
  float alpha,
  float const *A,
  int lda,
  float const *B,
  int ldb,
  float beta,
  float *C,
  int ldc,
  hipStream_t stream) {

  // Define type definition for single-precision CUTLASS GEMM with column-major
  // input matrices and 128x128x8 threadblock tile size (chosen by default).
  //
  // To keep the interface manageable, several helpers are defined for plausible compositions
  // including the following example for single-precision GEMM. Typical values are used as
  // default template arguments. See `cutlass/gemm/device/default_gemm_configuration.h` for more details.
  //
  // To view the full gemm device API interface, see `cutlass/gemm/device/gemm.h`

//   using ColumnMajor = cutlass::layout::ColumnMajor;

//   using CutlassGemm = cutlass::gemm::device::Gemm<float,        // Data-type of A matrix
//                                                   ColumnMajor,  // Layout of A matrix
//                                                   float,        // Data-type of B matrix
//                                                   ColumnMajor,  // Layout of B matrix
//                                                   float,        // Data-type of C matrix
//                                                   ColumnMajor>; // Layout of C matrix


///////////////////////////////////////////////////////////////////////////////////////////////////

// The code section below describes datatype for input, output matrices and computation between
// elements in input matrices.
using ElementAccumulator = float;                   // <- data type of accumulator
using ElementComputeEpilogue = ElementAccumulator;  // <- data type of epilogue operations
using ElementInputA = float;                        // <- data type of elements in input matrix A
using ElementInputB = float;                        // <- data type of elements in input matrix B
using ElementOutput = float;                        // <- data type of elements in output matrix D

// The code section below describes matrix layout of input and output matrices. Column Major for
// Matrix A, Row Major for Matrix B and Row Major for Matrix C
// using LayoutInputA = cutlass::layout::RowMajor;
// using LayoutInputB = cutlass::layout::ColumnMajor;
// using LayoutOutput = cutlass::layout::RowMajor;

using LayoutInputA = cutlass::layout::ColumnMajor;
using LayoutInputB = cutlass::layout::ColumnMajor;
using LayoutOutput = cutlass::layout::ColumnMajor;
// This code section describes whether you want to use tensor cores or regular SIMT cores on GPU SM
using MMAOp = cutlass::arch::OpClassTensorOp;

// This code section describes CUDA SM architecture number
using SmArch = cutlass::arch::Sm80;

// This code section describes the tile size a thread block will compute
using ShapeMMAThreadBlock =
    cutlass::gemm::GemmShape<256, 128, 16>;  // <- threadblock tile M = 256, N = 128, K = 16
// This code section describes tile size a warp will compute
using ShapeMMAWarp = cutlass::gemm::GemmShape<64, 64, 16>;  // <- warp tile M = 64, N = 64, K = 16
// This code section describes the size of MMA op
using ShapeMMAOp = cutlass::gemm::GemmShape<16, 8, 8>;  // <- MMA Op tile M = 16, N = 8, K = 8

// This code section describes how threadblocks are scheduled on GPU
using SwizzleThreadBlock = cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>;  // <- ??

// This code section describes the epilogue part of the kernel
using EpilogueOp = cutlass::epilogue::thread::LinearCombination<
    ElementOutput,                                     // <- data type of output matrix
    128 / cutlass::sizeof_bits<ElementOutput>::value,  // <- the number of elements per vectorized
                                                       // memory access. For a byte, it's 16
                                                       // elements. This becomes the vector width of
                                                       // math instructions in the epilogue too
    ElementAccumulator,                                // <- data type of accumulator
    ElementComputeEpilogue>;  // <- data type for alpha/beta in linear combination function

// Number of pipelines you want to use
constexpr int NumStages = 4;

using CutlassGemm = cutlass::gemm::device::Gemm<ElementInputA,
                                         LayoutInputA,
                                         ElementInputB,
                                         LayoutInputB,
                                         ElementOutput,
                                         LayoutOutput,
                                         ElementAccumulator,
                                         MMAOp,
                                         SmArch,
                                         ShapeMMAThreadBlock,
                                         ShapeMMAWarp,
                                         ShapeMMAOp,
                                         EpilogueOp,
                                         SwizzleThreadBlock,
                                         NumStages>;

  // Define a CUTLASS GEMM type
  CutlassGemm gemm_operator;

  // Construct the CUTLASS GEMM arguments object.
  //
  // One of CUTLASS's design patterns is to define gemm argument objects that are constructible
  // in host code and passed to kernels by value. These may include pointers, strides, scalars,
  // and other arguments needed by Gemm and its components.
  //
  // The benefits of this pattern are (1.) a structured, composable strategy for passing host-constructible
  // arguments to kernels and (2.) minimized initialization overhead on kernel entry.
  //
  CutlassGemm::Arguments args({M , N, K},  // Gemm Problem dimensions
                              {A, lda},    // Tensor-ref for source matrix A
                              {B, ldb},    // Tensor-ref for source matrix B
                              {C, ldc},    // Tensor-ref for source matrix C
                              {C, ldc},    // Tensor-ref for destination matrix D (may be different memory than source C matrix)
                              {alpha, beta}); // Scalars used in the Epilogue

  // Using the arguments, query for extra workspace required for matrix multiplication computation
  size_t workspace_size = CutlassGemm::get_workspace_size(args);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  //
  // Launch the CUTLASS GEMM kernel.
  //
  
  cutlass::Status status = gemm_operator(args, workspace.get(), stream);

  //
  // Return a hipError_t if the CUTLASS GEMM operator returned an error code.
  //

  if (status != cutlass::Status::kSuccess) {
    return hipErrorUnknown;
  }

  // Return success, if no errors were encountered.
  return hipSuccess;
}

hipError_t CublasSgemm(
  int M,
  int N,
  int K,
  float alpha,
  float const *A,
  int lda,
  float const *B,
  int ldb,
  float beta,
  float *C,
  int ldc,
  hipStream_t stream) {

    hipblasHandle_t handle;
    checkCudaErrors(hipblasCreate(&handle));
    checkCudaErrors(hipblasSetStream(handle, stream));
    checkCudaErrors(hipblasSgemm(handle,
                    HIPBLAS_OP_N, HIPBLAS_OP_N,
                    M, N, K,
                    &alpha,
                    A, lda,
                    B, ldb,
                    &beta,
                    C, ldc));

  hipError_t result = hipSuccess;
  return result;
  }


///////////////////////////////////////////////////////////////////////////////////////////////////
//
// The source code after this point in the file is generic CUDA using the CUDA Runtime API
// and simple CUDA kernels to initialize matrices and compute the general matrix product.
//
///////////////////////////////////////////////////////////////////////////////////////////////////

/// Kernel to initialize a matrix with small integers.
__global__ void InitializeMatrix_kernel(
  float *matrix,
  int rows,
  int columns,
  int seed = 0) {

  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;

  if (i < rows && j < columns) {
    int offset = i + j * rows;

    // Generate arbitrary elements.
    int const k = 16807;
    int const m = 16;
    float value = float(((offset + seed) * k % m) - m / 2);

    matrix[offset] = value;
  }
}

/// Simple function to initialize a matrix to arbitrary small integers.
hipError_t InitializeMatrix(float *matrix, int rows, int columns, int seed = 0) {

  dim3 block(16, 16);
  dim3 grid(
    (rows + block.x - 1) / block.x,
    (columns + block.y - 1) / block.y
  );

  InitializeMatrix_kernel<<< grid, block >>>(matrix, rows, columns, seed);

  return hipGetLastError();
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Allocates device memory for a matrix then fills with arbitrary small integers.
hipError_t AllocateMatrix(float **matrix, int rows, int columns, int seed = 0) {
  hipError_t result;

  size_t sizeof_matrix = sizeof(float) * rows * columns;

  // Allocate device memory.
  result = hipMalloc(reinterpret_cast<void **>(matrix), sizeof_matrix);

  if (result != hipSuccess) {
    std::cerr << "Failed to allocate matrix: "
      << hipGetErrorString(result) << std::endl;
    return result;
  }

  // Clear the allocation.
  result = hipMemset(*matrix, 0, sizeof_matrix);

  if (result != hipSuccess) {
    std::cerr << "Failed to clear matrix device memory: "
      << hipGetErrorString(result) << std::endl;
    return result;
  }

  // Initialize matrix elements to arbitrary small integers.
  result = InitializeMatrix(*matrix, rows, columns, seed);

  if (result != hipSuccess) {
    std::cerr << "Failed to initialize matrix: "
      << hipGetErrorString(result) << std::endl;
    return result;
  }

  return result;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Naive reference GEMM computation.
__global__ void ReferenceGemm_kernel(
  int M,
  int N,
  int K,
  float alpha,
  float const *A,
  int lda,
  float const *B,
  int ldb,
  float beta,
  float *C,
  int ldc) {

  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;

  if (i < M && j < N) {
    float accumulator = 0;

    for (int k = 0; k < K; ++k) {
      accumulator += A[i + k * lda] * B[k + j * ldb];
    }

    C[i + j * ldc] = alpha * accumulator + beta * C[i + j * ldc];
  }
}

/// Reference GEMM computation.
hipError_t ReferenceGemm(
  int M,
  int N,
  int K,
  float alpha,
  float const *A,
  int lda,
  float const *B,
  int ldb,
  float beta,
  float *C,
  int ldc) {

  dim3 block(16, 16);
  dim3 grid(
    (M + block.x - 1) / block.x,
    (N + block.y - 1) / block.y
  );

  ReferenceGemm_kernel<<< grid, block >>>(M, N, K, alpha, A, lda, B, ldb, beta, C, ldc);

  return hipGetLastError();
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Result structure
struct ResultS {

  double runtime_ms;
  double gflops;
  cutlass::Status status;
  hipError_t error;
  bool passed;

  //
  // Methods
  //

  ResultS(
    double runtime_ms = 0,
    double gflops = 0,
    cutlass::Status status = cutlass::Status::kSuccess,
    hipError_t error = hipSuccess
  ):
    runtime_ms(runtime_ms), gflops(gflops), status(status), error(error), passed(true) { }
};

/// Compute performance in GFLOP/s
double gflops(double runtime_s, int M, int N, int K, int iterations) {

// Number of real-valued multiply-adds 
int64_t fmas = M * N * N * iterations;

// Two flops per multiply-add
return 2.0 * double(fmas) / double(1.0e9) / runtime_s;
}

/// Allocate several matrices in GPU device memory and call a single-precision
/// CUTLASS cuBLAS GEMM kernel.
hipError_t CompareGemm(int M, int N, int K, float alpha, float beta) {
  hipError_t result;

  //
  // Define several matrices to be used as operands to GEMM kernels.
  //

  // Compute leading dimensions for each matrix.
  int lda = M;
  int ldb = K;
  int ldc = M;

  // Compute size in bytes of the C matrix.
  size_t sizeof_C = sizeof(float) * ldc * N;

  // Define pointers to matrices in GPU device memory.
  float *A;
  float *B;
  float *C_cutlass;
  float *C_reference;

  float *A_p0;
  float *A_p1;
  float *B_p0;
  float *B_p1;
  float *C_cutlass_p0;
  float *C_cutlass_p1;

  //
  // Allocate matrices in GPU device memory with arbitrary seeds.
  //

  result = AllocateMatrix(&A, M, K, 0);

  if (result !=  hipSuccess) {
    return result;
  }

  result = AllocateMatrix(&B, K, N, 17);

  if (result !=  hipSuccess) {
    hipFree(A);
    return result;
  }

  result = AllocateMatrix(&C_cutlass, M, N, 101);

  if (result != hipSuccess) {
    hipFree(A);
    hipFree(B);
    return result;
  }

  result = AllocateMatrix(&C_reference, M, N, 101);

  if (result != hipSuccess) {
    hipFree(A);
    hipFree(B);
    hipFree(C_cutlass);
    return result;
  }

  result = hipMemcpy(C_reference, C_cutlass, sizeof_C, hipMemcpyDeviceToDevice);

  if (result != hipSuccess) {
    std::cerr << "Failed to copy C_cutlass matrix to C_reference: "
      << hipGetErrorString(result) << std::endl;

    hipFree(C_reference);
    hipFree(C_cutlass);
    hipFree(B);
    hipFree(A);

    return result;
  }

  result = AllocateMatrix(&A_p0, M, K, 66);
  result = AllocateMatrix(&A_p1, M, K, 66);
  result = AllocateMatrix(&B_p0, K, N/2, 66);
  result = AllocateMatrix(&B_p1, K, N/2, 66);
  result = AllocateMatrix(&C_cutlass_p0, M, N/2, 66);
  result = AllocateMatrix(&C_cutlass_p1, M, N/2, 66);

    hipStream_t stream0;
    hipStreamCreate(&stream0);
    hipStream_t stream1;
    hipStreamCreate(&stream1);

  ResultS result_s;

  // warmup
  result = CutlassSgemmNN(M, N/2, K, alpha, A_p0, lda, B_p0, ldb, beta, C_cutlass_p0, ldc, stream0);
  CublasSgemm(M, N, K, alpha, A, lda, B, ldb, beta, C_cutlass, ldc, stream0);

  //
  // Construct events
  //
  hipEvent_t events[2];

  for (auto & event : events) {
    result_s.error = hipEventCreate(&event);
    if (result_s.error != hipSuccess) {
      std::cerr << "hipEventCreate() failed: " << hipGetErrorString(result_s.error) << std::endl;
    }
  }

  // Record an event at the start of a series of GEMMs
  result_s.error = hipEventRecord(events[0]);
  if (result_s.error != hipSuccess) {
    std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result_s.error) << std::endl;
  }

  // Launch CUTLASS GEMM.
    int iterations = 10;
  for (int iter = 0; iter < iterations; ++iter) {
    result = CutlassSgemmNN(M, N/2, K, alpha, A_p0, lda, B_p0, ldb, beta, C_cutlass_p0, ldc, stream0);
    result = CutlassSgemmNN(M, N/2, K, alpha, A_p1, lda, B_p1, ldb, beta, C_cutlass_p1, ldc, stream1);
  }

  // Record an event when the GEMMs are complete
  result_s.error = hipEventRecord(events[1]);
  if (result_s.error != hipSuccess) {
    std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result_s.error) << std::endl;
  }

  // Wait for work on the device to complete.
  result_s.error = hipEventSynchronize(events[1]);
  if (result_s.error != hipSuccess) {
    std::cerr << "hipEventSynchronize() failed: " << hipGetErrorString(result_s.error) << std::endl;
  }

  // Measure elapsed runtime
  float runtime_ms = 0;
  result_s.error = hipEventElapsedTime(&runtime_ms, events[0], events[1]);
  if (result_s.error != hipSuccess) {
    std::cerr << "cudaEventElapsed() failed: " << hipGetErrorString(result_s.error) << std::endl;
  }

  // Compute average runtime and GFLOPs.
  result_s.runtime_ms = double(runtime_ms) / double(iterations);
  result_s.gflops = gflops(result_s.runtime_ms / 1000.0, M, N, K, iterations);
    std::cout << "CUTLASS Runtime: " << result_s.runtime_ms << " ms" << std::endl;
    std::cout << "CUTLASS GFLOPs: " << result_s.gflops << std::endl;


  ///////////////////////////////////////////////////////////////////////////////////////////////////
 // Record an event at the start of a series of GEMMs
  hipEventRecord(events[0]);

  // Launch cuBLAS GEMM.
  for (int iter = 0; iter < iterations; ++iter) {
    CublasSgemm(M, N, K, alpha, A, lda, B, ldb, beta, C_cutlass, ldc, stream0);
  }

  // Record an event when the GEMMs are complete
  hipEventRecord(events[1]);

  // Wait for work on the device to complete.
  hipEventSynchronize(events[1]);

  // Measure elapsed runtime
  runtime_ms = 0;
  hipEventElapsedTime(&runtime_ms, events[0], events[1]);

  // Compute average runtime and GFLOPs.
  double cublas_runtime_ms = double(runtime_ms) / double(iterations);
  double cublas_gflops = gflops(cublas_runtime_ms / 1000.0, M, N, K, iterations);
    std::cout << "cuBLAS Runtime: " << cublas_runtime_ms << " ms" << std::endl;
    std::cout << "cuBLAS GFLOPs: " << cublas_gflops << std::endl;

  // Cleanup
  for (auto event : events) {
    (void)hipEventDestroy(event);
  }

  if (result != hipSuccess) {
    std::cerr << "CUTLASS GEMM kernel failed: "
      << hipGetErrorString(result) << std::endl;

    hipFree(C_reference);
    hipFree(C_cutlass);
    hipFree(B);
    hipFree(A);

    return result;
  }

  //
  // Verify.
  //

  // Launch reference GEMM
  result = ReferenceGemm(M, N, K, alpha, A, lda, B, ldb, beta, C_reference, ldc);

  if (result != hipSuccess) {
    std::cerr << "Reference GEMM kernel failed: "
      << hipGetErrorString(result) << std::endl;

    hipFree(C_reference);
    hipFree(C_cutlass);
    hipFree(B);
    hipFree(A);

    return result;
  }

  // Copy to host and verify equivalence.
  std::vector<float> host_cutlass(ldc * N, 0);
  std::vector<float> host_reference(ldc * N, 0);

  result = hipMemcpy(host_cutlass.data(), C_cutlass, sizeof_C, hipMemcpyDeviceToHost);

  if (result != hipSuccess) {
    std::cerr << "Failed to copy CUTLASS GEMM results: "
      << hipGetErrorString(result) << std::endl;

    hipFree(C_reference);
    hipFree(C_cutlass);
    hipFree(B);
    hipFree(A);

    return result;
  }

  result = hipMemcpy(host_reference.data(), C_reference, sizeof_C, hipMemcpyDeviceToHost);

  if (result != hipSuccess) {
    std::cerr << "Failed to copy Reference GEMM results: "
      << hipGetErrorString(result) << std::endl;

    hipFree(C_reference);
    hipFree(C_cutlass);
    hipFree(B);
    hipFree(A);

    return result;
  }

  //
  // Free device memory allocations.
  //

  hipFree(C_reference);
  hipFree(C_cutlass);
  hipFree(B);
  hipFree(A);

  //
  // Test for bit equivalence of results.
  //

  if (host_cutlass != host_reference) {
    std::cerr << "CUTLASS results incorrect." << std::endl;

    return hipErrorUnknown;
  }

  return hipSuccess;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Entry point to basic_gemm example.
//
// usage:
//
//   00_basic_gemm <M> <N> <K> <alpha> <beta>
//
int main(int argc, const char *arg[]) {

  //
  // Parse the command line to obtain GEMM dimensions and scalar values.
  //

  // GEMM problem dimensions.
//   int problem[3] = { 128, 128, 128 };
//   int problem[3] = { 2048, 256, 2048 };
//   int problem[3] = { 4096, 256, 4096 };
  int problem[3] = { 4096, 256, 2048 };

  for (int i = 1; i < argc && i < 4; ++i) {
    std::stringstream ss(arg[i]);
    ss >> problem[i - 1];
  }

  // Scalars used for linear scaling the result of the matrix product.
  float scalars[2] = { 1, 0 };

  for (int i = 4; i < argc && i < 6; ++i) {
    std::stringstream ss(arg[i]);
    ss >> scalars[i - 4];
  }

  //
  // Run the CUTLASS GEMM test.
  //

  hipError_t result = CompareGemm(
    problem[0],     // GEMM M dimension
    problem[1],     // GEMM N dimension
    problem[2],     // GEMM K dimension
    scalars[0],     // alpha
    scalars[1]      // beta
  );

  if (result == hipSuccess) {
    std::cout << "Passed." << std::endl;
  }

  // Exit.
  return result == hipSuccess ? 0 : -1;
}

///////////////////////////////////////////////////////////////////////////////////////////////////
